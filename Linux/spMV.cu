#include "hip/hip_runtime.h"
#include "spMV.cuh"

/* Perform preconditioner matrix vector multiplication - a stream kernel with 2X improvement over cusparse APi */
__global__ void spMV_M_kernel(const double* d_M, const double* d_V, double* d_target)

{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < d_Nnode)
	{
		d_target[tid] = d_M[tid] * d_V[tid];
	}

}


/* Perform large global uncoalsed sparse matrix vector multiplication with summed spMV output */
template <int blockDIM> __global__
void spMV_sparse(const double* __restrict__ a_expand,
	const int* __restrict__ ja_expand,
	const double* __restrict__ v,
	double* __restrict__ v_expand,
	double* __restrict__ spMV)
{
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int lane = tid & 31; // lane ID within a warp

	__shared__ volatile double cache[blockDIM];

	if (tid < d_Nnode * 16)
	{
		v_expand[tid] = v[ja_expand[tid]];
		cache[threadIdx.x] = a_expand[tid] * v_expand[tid];
		// __syncthreads(); // not necessary since the reduction is done within the warp

		// in warp reduction

		if (lane < 8 || (lane > 15 && lane < 24)) cache[threadIdx.x] += cache[threadIdx.x + 8];
		if (lane < 4 || (lane > 15 && lane < 20)) cache[threadIdx.x] += cache[threadIdx.x + 4];
		if (lane < 2 || (lane > 15 && lane < 18)) cache[threadIdx.x] += cache[threadIdx.x + 2];
		if (lane < 1 || (lane > 15 && lane < 17)) cache[threadIdx.x] += cache[threadIdx.x + 1];

		if (lane == 0 || lane == 16) spMV[tid / 16] = cache[threadIdx.x];

	}

}


/* spMV kernel with one thread per row - comparable with hipsparseSpMV APi */
__global__ void spMV_thread_kernel(const int* d_ia, const int* d_ja, const double* d_val, const double* x, double* y)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if (row < d_Nnode) {
		double dot = 0;
		int row_start = d_ia[row];
		int row_end = d_ia[row + 1];
		for (int jj = row_start; jj < row_end; jj++)
			dot += d_val[jj] * x[d_ja[jj]];
		y[row] += dot;
	}
}

/* spMV kernel with one warp (32 thread) per row - poor performance */
template <int blockDIM> __global__
void spMV_warp_kernel(const int* d_ia, const int* d_ja, const double* d_val, const double* x, double* y)
{
	__shared__ double vals[blockDIM];

	int thread_id = blockDim.x * blockIdx.x + threadIdx.x; // global thread index 
	int warp_id = thread_id / 32; // global warp index 
	int lane = thread_id & (32 - 1); // one warp per row 
	int row = warp_id;
	if (row < d_Nnode)
	{
		int row_start = d_ia[row];
		int row_end = d_ia[row + 1];// compute running sum per thread 
		vals[threadIdx.x] = 0;

		for (int jj = row_start + lane; jj < row_end; jj += 32)
			vals[threadIdx.x] += d_val[jj] * x[d_ja[jj]];// parallel reduction in shared memory 

		if (lane < 16) vals[threadIdx.x] += vals[threadIdx.x + 16];
		if (lane < 8) vals[threadIdx.x] += vals[threadIdx.x + 8];
		if (lane < 4) vals[threadIdx.x] += vals[threadIdx.x + 4];
		if (lane < 2) vals[threadIdx.x] += vals[threadIdx.x + 2];
		if (lane < 1) vals[threadIdx.x] += vals[threadIdx.x + 1];

		if (lane == 0) y[row] += vals[threadIdx.x]; // first thread writes the result 
	}
}

/* Fully looped dot product kernal with shared memory function - poor performance */
__global__ void dot_product_kernel(const double* __restrict__ x, const double* __restrict__ y, double* __restrict__ dot)
{
	unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;

	extern __shared__ double cache[];

	cache[threadIdx.x] = x[index] * y[index];

	__syncthreads();

	// reduction
	unsigned int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			cache[threadIdx.x] += cache[threadIdx.x + i];
		}
		__syncthreads();
		i /= 2;
	}


	if (threadIdx.x == 0) {
		atomicAdd(dot, cache[0]);
	}
}

/* Fully unrolled templated dot product with shared memory function - best performance so far
	Runtime performance beats hipblasDdot api on Nsight compute in release mode
	hipblasDdot api performance is invariant to debug/release mode, which means the optimization is already done by Nvidia build-in api function
*/
template <int blockDIM> __global__
void dot_product_kernel_unroll(const double* __restrict__ x, const double* __restrict__ y, double* __restrict__ dot)
{
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x * 8; // block unroll factor of 8

	__shared__ volatile double cache[blockDIM]; // dynamically allocate the shared memory according to block size

	double temp = 0.0;

	if (tid + 7 * blockDim.x <= d_Nnode)
	{
		double a1 = x[tid] * y[tid];
		double a2 = x[tid + blockDim.x] * y[tid + blockDim.x];
		double a3 = x[tid + 2 * blockDim.x] * y[tid + 2 * blockDim.x];
		double a4 = x[tid + 3 * blockDim.x] * y[tid + 3 * blockDim.x];
		double a5 = x[tid + 4 * blockDim.x] * y[tid + 4 * blockDim.x];
		double a6 = x[tid + 5 * blockDim.x] * y[tid + 5 * blockDim.x];
		double a7 = x[tid + 6 * blockDim.x] * y[tid + 6 * blockDim.x];
		double a8 = x[tid + 7 * blockDim.x] * y[tid + 7 * blockDim.x];

		temp = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
	}
	else
	{
		unsigned int Grid_num = d_Nnode / (blockDim.x * 8); // integer number of assigned grid with unroll factor of 8

		tid = threadIdx.x + (Grid_num * blockDim.x * 8 + blockDim.x * (blockIdx.x - Grid_num)); // Remaining block will do the dot product without any unroll factor

		if (tid < d_Nnode) temp = x[tid] * y[tid];
	}

	cache[threadIdx.x] = temp;

	__syncthreads();

	// in-place reduction in shared memory
	if (blockDIM >= 1024 && threadIdx.x < 512) cache[threadIdx.x] += cache[threadIdx.x + 512];
	__syncthreads();

	if (blockDIM >= 512 && threadIdx.x < 256) cache[threadIdx.x] += cache[threadIdx.x + 256];
	__syncthreads();

	if (blockDIM >= 256 && threadIdx.x < 128) cache[threadIdx.x] += cache[threadIdx.x + 128];
	__syncthreads();

	if (blockDIM >= 128 && threadIdx.x < 64) cache[threadIdx.x] += cache[threadIdx.x + 64];
	__syncthreads();

	// unrolling warp
	if (threadIdx.x < 32)
	{
		cache[threadIdx.x] += cache[threadIdx.x + 32];
		cache[threadIdx.x] += cache[threadIdx.x + 16];
		cache[threadIdx.x] += cache[threadIdx.x + 8];
		cache[threadIdx.x] += cache[threadIdx.x + 4];
		cache[threadIdx.x] += cache[threadIdx.x + 2];
		cache[threadIdx.x] += cache[threadIdx.x + 1];
	}

	if (threadIdx.x == 0) {
		atomicAdd(dot, cache[0]);
	}
}

/*
	Two dot product operations in one kernel launch
*/
template <int blockDIM> __global__
void dot_product_kernel_V2_unroll(const double* __restrict__ x, const double* __restrict__ y, double* __restrict__ sum, double* __restrict__ dot)
{
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x * 8; // block unroll factor of 8

	__shared__ volatile double cache[blockDIM]; // dynamically allocate the shared memory according to block size

	double temp_sum = 0.0;
	double temp_dot = 0.0;

	// x*x dot product

	if (tid + 7 * blockDim.x <= d_Nnode)
	{
		double a1 = x[tid] * x[tid];
		double a11 = x[tid] * y[tid];

		double a2 = x[tid + blockDim.x] * x[tid + blockDim.x];
		double a22 = x[tid + blockDim.x] * y[tid + blockDim.x];

		double a3 = x[tid + 2 * blockDim.x] * x[tid + 2 * blockDim.x];
		double a33 = x[tid + 2 * blockDim.x] * y[tid + 2 * blockDim.x];

		double a4 = x[tid + 3 * blockDim.x] * x[tid + 3 * blockDim.x];
		double a44 = x[tid + 3 * blockDim.x] * y[tid + 3 * blockDim.x];

		double a5 = x[tid + 4 * blockDim.x] * x[tid + 4 * blockDim.x];
		double a55 = x[tid + 4 * blockDim.x] * y[tid + 4 * blockDim.x];

		double a6 = x[tid + 5 * blockDim.x] * x[tid + 5 * blockDim.x];
		double a66 = x[tid + 5 * blockDim.x] * y[tid + 5 * blockDim.x];

		double a7 = x[tid + 6 * blockDim.x] * x[tid + 6 * blockDim.x];
		double a77 = x[tid + 6 * blockDim.x] * y[tid + 6 * blockDim.x];

		double a8 = x[tid + 7 * blockDim.x] * x[tid + 7 * blockDim.x];
		double a88 = x[tid + 7 * blockDim.x] * y[tid + 7 * blockDim.x];

		temp_sum = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;

		temp_dot = a11 + a22 + a33 + a44 + a55 + a66 + a77 + a88;
	}
	else
	{
		unsigned int Grid_num = d_Nnode / (blockDim.x * 8); // integer number of assigned grid with unroll factor of 8

		tid = threadIdx.x + (Grid_num * blockDim.x * 8 + blockDim.x * (blockIdx.x - Grid_num)); // Remaining block will do the dot product without any unroll factor

		if (tid < d_Nnode) temp_sum = x[tid] * x[tid]; temp_dot = x[tid] * y[tid];
	}

	// x*y dot product

	cache[threadIdx.x] = temp_sum;

	__syncthreads();

	// in-place reduction in shared memory
	if (blockDIM >= 1024 && threadIdx.x < 512) cache[threadIdx.x] += cache[threadIdx.x + 512];
	__syncthreads();

	if (blockDIM >= 512 && threadIdx.x < 256) cache[threadIdx.x] += cache[threadIdx.x + 256];
	__syncthreads();

	if (blockDIM >= 256 && threadIdx.x < 128) cache[threadIdx.x] += cache[threadIdx.x + 128];
	__syncthreads();

	if (blockDIM >= 128 && threadIdx.x < 64) cache[threadIdx.x] += cache[threadIdx.x + 64];
	__syncthreads();

	// unrolling warp
	if (threadIdx.x < 32)
	{
		cache[threadIdx.x] += cache[threadIdx.x + 32];
		cache[threadIdx.x] += cache[threadIdx.x + 16];
		cache[threadIdx.x] += cache[threadIdx.x + 8];
		cache[threadIdx.x] += cache[threadIdx.x + 4];
		cache[threadIdx.x] += cache[threadIdx.x + 2];
		cache[threadIdx.x] += cache[threadIdx.x + 1];
	}

	if (threadIdx.x == 0) {
		atomicAdd(sum, cache[0]);
	}

	cache[threadIdx.x] = temp_dot;

	__syncthreads();

	// in-place reduction in shared memory
	if (blockDIM >= 1024 && threadIdx.x < 512) cache[threadIdx.x] += cache[threadIdx.x + 512];
	__syncthreads();

	if (blockDIM >= 512 && threadIdx.x < 256) cache[threadIdx.x] += cache[threadIdx.x + 256];
	__syncthreads();

	if (blockDIM >= 256 && threadIdx.x < 128) cache[threadIdx.x] += cache[threadIdx.x + 128];
	__syncthreads();

	if (blockDIM >= 128 && threadIdx.x < 64) cache[threadIdx.x] += cache[threadIdx.x + 64];
	__syncthreads();

	// unrolling warp
	if (threadIdx.x < 32)
	{
		cache[threadIdx.x] += cache[threadIdx.x + 32];
		cache[threadIdx.x] += cache[threadIdx.x + 16];
		cache[threadIdx.x] += cache[threadIdx.x + 8];
		cache[threadIdx.x] += cache[threadIdx.x + 4];
		cache[threadIdx.x] += cache[threadIdx.x + 2];
		cache[threadIdx.x] += cache[threadIdx.x + 1];
	}

	if (threadIdx.x == 0) {
		atomicAdd(dot, cache[0]);
	}
}

/*  y = a*x + y
	similar performance of hipblasDaxpy, can be used inter-changeably
*/
__global__ void axpy_kernal(const double* __restrict__ x, double* __restrict__ y)
{
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < d_Nnode)
	{
		y[tid] += d_scale * x[tid];
	}
}

/*
	Two axpy operations in one function call:
	1. y1 += scale * x1
	2. y2 -= scale * x2
*/
__global__ void axpy_kernal_V2(const double* __restrict__ x1, double* __restrict__ y1, const double* __restrict__ x2, double* __restrict__ y2)
{
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < d_Nnode)
	{
		y1[tid] += d_scale * x1[tid];
		y2[tid] -= d_scale * x2[tid];
	}
}

/*
	This kernel combines the operation for P_{j+1} = r_{j+1} + beta_j * (P_j - omega_j*AP_j)
*/
__global__ void p_update_kernel(double* __restrict__ P, const double* __restrict__ AP, const double* __restrict__ r)
{
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < d_Nnode)
	{
		P[tid] = r[tid] + d_beta * (P[tid] - d_omega * AP[tid]);
	}
}

/*
	More than 2 - 3X improvement over cublas APi due to removal of multi-mode computation
*/
template <int blockDIM> __global__
void nrm2_kernel_unroll(const double* __restrict__ x, double* __restrict__ sum)
{
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x * 8; // block unroll factor of 8

	__shared__ volatile double cache[blockDIM]; // dynamically allocate the shared memory according to block size

	double temp = 0.0;

	if (tid + 7 * blockDim.x <= d_Nnode)
	{
		double a1 = x[tid] * x[tid];
		double a2 = x[tid + blockDim.x] * x[tid + blockDim.x];
		double a3 = x[tid + 2 * blockDim.x] * x[tid + 2 * blockDim.x];
		double a4 = x[tid + 3 * blockDim.x] * x[tid + 3 * blockDim.x];
		double a5 = x[tid + 4 * blockDim.x] * x[tid + 4 * blockDim.x];
		double a6 = x[tid + 5 * blockDim.x] * x[tid + 5 * blockDim.x];
		double a7 = x[tid + 6 * blockDim.x] * x[tid + 6 * blockDim.x];
		double a8 = x[tid + 7 * blockDim.x] * x[tid + 7 * blockDim.x];

		temp = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
	}
	else
	{
		unsigned int Grid_num = d_Nnode / (blockDim.x * 8); // integer number of assigned grid with unroll factor of 8

		tid = threadIdx.x + (Grid_num * blockDim.x * 8 + blockDim.x * (blockIdx.x - Grid_num)); // Remaining block will do the dot product without any unroll factor

		if (tid < d_Nnode) temp = x[tid] * x[tid];
	}

	cache[threadIdx.x] = temp;

	__syncthreads();

	// in-place reduction in shared memory
	if (blockDIM >= 1024 && threadIdx.x < 512) cache[threadIdx.x] += cache[threadIdx.x + 512];
	__syncthreads();

	if (blockDIM >= 512 && threadIdx.x < 256) cache[threadIdx.x] += cache[threadIdx.x + 256];
	__syncthreads();

	if (blockDIM >= 256 && threadIdx.x < 128) cache[threadIdx.x] += cache[threadIdx.x + 128];
	__syncthreads();

	if (blockDIM >= 128 && threadIdx.x < 64) cache[threadIdx.x] += cache[threadIdx.x + 64];
	__syncthreads();

	// unrolling warp
	if (threadIdx.x < 32)
	{

		cache[threadIdx.x] += cache[threadIdx.x + 32];
		cache[threadIdx.x] += cache[threadIdx.x + 16];
		cache[threadIdx.x] += cache[threadIdx.x + 8];
		cache[threadIdx.x] += cache[threadIdx.x + 4];
		cache[threadIdx.x] += cache[threadIdx.x + 2];
		cache[threadIdx.x] += cache[threadIdx.x + 1];

	}

	if (threadIdx.x == 0) {
		atomicAdd(sum, cache[0]);
	}
}


namespace cuBLAS {

	/* Return matrix X vector = d_Target vector on device */
	void spMV_M(dim3& Grid, dim3& Block, const double* d_M, const double* d_V, double* d_Target)
	{

		spMV_M_kernel << <Grid, Block >> > (d_M, d_V, d_Target);

	}

	void spMV(dim3& Grid, dim3& Block,
		const double* d_a_expand,
		const int* d_ja_expand,
		const double* d_v, double* d_v_expanded, double* d_spMV)
	{
		switch (Block.x)
		{
		case 1024:
			spMV_sparse <1024> << <Grid, Block >> > (d_a_expand, d_ja_expand, d_v, d_v_expanded, d_spMV);
			break;
		case 512:
			spMV_sparse <512> << <Grid, Block >> > (d_a_expand, d_ja_expand, d_v, d_v_expanded, d_spMV);
			break;
		case 256:
			spMV_sparse <256> << <Grid, Block >> > (d_a_expand, d_ja_expand, d_v, d_v_expanded, d_spMV);
			break;
		}
	}

	void spMV_thread(dim3& Grid, dim3& Block,
		const int* d_ia_expand,
		const int* d_ja_expand,
		const double* d_a,
		const double* d_v,
		double* d_spMV)
	{
		checkCudaErrors(hipMemset(d_spMV, 0, Nnode * sizeof(double)));
		spMV_thread_kernel << <Grid, Block >> > (d_ia_expand, d_ja_expand, d_a, d_v, d_spMV);
	}

	void spMV_warp(dim3& Grid, dim3& Block,
		const int* d_ia_expand,
		const int* d_ja_expand,
		const double* d_a,
		const double* d_v,
		double* d_spMV)
	{

		checkCudaErrors(hipMemset(d_spMV, 0, Nnode * sizeof(double)));

		switch (Block.x)
		{
		case 1024:
			spMV_warp_kernel <1024> << <Grid, Block >> > (d_ia_expand, d_ja_expand, d_a, d_v, d_spMV);
			break;
		case 512:
			spMV_warp_kernel <512> << <Grid, Block >> > (d_ia_expand, d_ja_expand, d_a, d_v, d_spMV);
			break;
		case 256:
			spMV_warp_kernel <256> << <Grid, Block >> > (d_ia_expand, d_ja_expand, d_a, d_v, d_spMV);
			break;
		}

	}

	/* Return the dot product of vector 1 (V_1) and vector 2 (V_2) to a host pointer - product */
	void dot_product(dim3& Grid, dim3& Block, const double* __restrict__ d_V_1, const double* __restrict__ d_V_2, double* __restrict__ product, double* __restrict__ d_product)
	{

		int Grid_unrolled = (Grid.x - 8 * (Grid.x / 8)) + Grid.x / 8;

		checkCudaErrors(hipMemset(d_product, 0, sizeof(double))); // Initialize to 0

		switch (Block.x)
		{

		case 1024:
			dot_product_kernel_unroll <1024> << <Grid_unrolled, Block >> > (d_V_1, d_V_2, d_product);
			break;

		case 512:
			dot_product_kernel_unroll <512> << <Grid_unrolled, Block >> > (d_V_1, d_V_2, d_product);
			break;

		case 256:
			dot_product_kernel_unroll <256> << <Grid_unrolled, Block >> > (d_V_1, d_V_2, d_product);
			break;

		case 128:
			dot_product_kernel_unroll <128> << <Grid_unrolled, Block >> > (d_V_1, d_V_2, d_product);
			break;

		}

		checkCudaErrors(hipMemcpy(product, d_product, sizeof(double), hipMemcpyDeviceToHost));

	}

	/* Return the dot product of vector 1 - vectort 1 (V_1) and vector 1 - vector 2 (V_2) to a host pointer - product */
	void dot_product_V2(dim3& Grid, dim3& Block, const double* __restrict__ d_V_1, const double* __restrict__ d_V_2,
		double* __restrict__ sum, double* __restrict__ d_sum, double* __restrict__ dot, double* __restrict__ d_dot)
	{

		int Grid_unrolled = (Grid.x - 8 * (Grid.x / 8)) + Grid.x / 8;

		checkCudaErrors(hipMemset(d_sum, 0, sizeof(double))); // Initialize to 0
		checkCudaErrors(hipMemset(d_dot, 0, sizeof(double))); // Initialize to 0

		switch (Block.x)
		{

		case 1024:
			dot_product_kernel_V2_unroll <1024> << <Grid_unrolled, Block >> > (d_V_1, d_V_2, d_sum, d_dot);
			break;

		case 512:
			dot_product_kernel_V2_unroll <512> << <Grid_unrolled, Block >> > (d_V_1, d_V_2, d_sum, d_dot);
			break;

		case 256:
			dot_product_kernel_V2_unroll <256> << <Grid_unrolled, Block >> > (d_V_1, d_V_2, d_sum, d_dot);
			break;

		case 128:
			dot_product_kernel_V2_unroll <128> << <Grid_unrolled, Block >> > (d_V_1, d_V_2, d_sum, d_dot);
			break;

		}

		checkCudaErrors(hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(dot, d_dot, sizeof(double), hipMemcpyDeviceToHost));

	}

	/* Return the nrm2 product of vector V to a host pointer - sum */
	void nrm2(dim3& Grid, dim3& Block, const double* __restrict__ V, double* __restrict__ sum, double* __restrict__ d_sum)
	{

		int Grid_unrolled = (Grid.x - 8 * (Grid.x / 8)) + Grid.x / 8;

		checkCudaErrors(hipMemset(d_sum, 0, sizeof(double))); // Initialize to 0

		switch (Block.x)
		{

		case 1024:
			nrm2_kernel_unroll <1024> << <Grid_unrolled, Block >> > (V, d_sum);
			break;

		case 512:
			nrm2_kernel_unroll <512> << <Grid_unrolled, Block >> > (V, d_sum);
			break;

		case 256:
			nrm2_kernel_unroll <256> << <Grid_unrolled, Block >> > (V, d_sum);
			break;

		case 128:
			nrm2_kernel_unroll <128> << <Grid_unrolled, Block >> > (V, d_sum);
			break;

		}

		checkCudaErrors(hipMemcpy(sum, d_sum, sizeof(double), hipMemcpyDeviceToHost));

		sum[0] = sqrt(sum[0]); // doing the square root on the host side

	}

	/* Return the vector product of vector P_{j+1} = r_{j+1} + beta_j * (P_j - omega_j*AP_j) */
	void p_update(dim3& Grid, dim3& Block, double* __restrict__ P, const double* __restrict__ AP, const double* __restrict__ r, const double& omega, const double& beta)
	{

		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &omega, sizeof(double))); // get the constant omega factor
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_beta), &beta, sizeof(double))); // get the constant beta factor

		p_update_kernel << <Grid, Block >> > (P, AP, r);

	}

	/* Return the vector product of vector y = y + scale * x */
	void axpy(dim3& Grid, dim3& Block, const double* __restrict__ x, double* __restrict__ y, const double& scale)
	{

		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_scale), &scale, sizeof(double))); // get the constant scale factor

		axpy_kernal << <Grid, Block >> > (x, y);

	}

	/* Return the 2 vector product of vector y1 = y1 + scale * x1 | y2 = y2 - scale * x2 */
	void axpy_V2(dim3& Grid, dim3& Block, const double* __restrict__ x1, double* __restrict__ y1, const double* __restrict__ x2, double* __restrict__ y2, const double& scale)
	{

		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_scale), &scale, sizeof(double))); // get the constant scale factor

		axpy_kernal_V2 << <Grid, Block >> > (x1, y1, x2, y2);

	}

	void get_const_int_symbol(const int& h_symbol)
	{
		/*
		int* ptr;
		checkCudaErrors(hipGetSymbolAddress((void**)&ptr, d_Nnode)); // On device side, it doesn't really care what is the type that the pointer is pointing to
																	  // It only needs a void pointer to allocate the memory space.
		*/
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_Nnode), &h_symbol, sizeof(int))); // For device variable, you can also assign a void* to any type variable (e.g int/double/double)
		// For host variable, however, you can only assign pointer to array or pointer type variable.
		// & operator is needed for host variable to be assigned by pointer
	}
}
